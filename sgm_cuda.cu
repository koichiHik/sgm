

// cuda
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>

// STL
#include <numeric>

// sgm
#include <disparity_method.h>

// Original
#include <sgm_cuda.h>

namespace stereo {

SGMCuda::SGMCuda() : p1(100), p2(100), finishCalled(false) {}

SGMCuda::~SGMCuda() {
  if (!finishCalled) {
    finish_disparity_method();
    finishCalled = true;
  }
}

bool SGMCuda::initialize(uint8_t p1, uint8_t p2) {
  init_disparity_method(p1, p2);
  finishCalled = false;
  return true;
}

void SGMCuda::generateDisparityMat(const cv::Mat &leftSrc,
                                   const cv::Mat &rightSrc, cv::Mat &disp) {
  float elapsedTime;

  cv::Mat left = leftSrc.clone();
  cv::Mat right = rightSrc.clone();

  if (1 < left.channels()) {
    cv::cvtColor(left, left, CV_RGB2GRAY);
  }

  if (1 < right.channels()) {
    cv::cvtColor(right, right, CV_RGB2GRAY);
  }

  cv::Mat dispMat = compute_disparity_method(left, right, &elapsedTime, "", "");

  {
    static int cnt = 0;
    spentTime.push_back(elapsedTime);
    cnt++;

    if (VEC_SIZE <= cnt) {
      cnt = 0;
    }
  }

  dispMat.copyTo(disp);
}

bool SGMCuda::end() {
  if (!finishCalled) {
    finish_disparity_method();
    finishCalled = true;
  }
  return true;
}

double SGMCuda::getSpentTimeForCur100Frames() {
  return std::accumulate(spentTime.begin(), spentTime.end(), 0.0) /
         (double)(spentTime.size());
}
}